#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>




# /*
# N - batch size of the input
# C - channel size of input
# K - kernel height = kernel width
# M - no of output feature maps( output channel size )
# H - height of the input data
# W - width of the input data
# P - padding ( heigth  = width ) , P/2 units added on the either sides
# S - stride ( height = width) , S units in one direction
# H_out - height of output data
# W_out  - weight ofoutput data

# - each thread will compute one element of one output feature map

# */

__global__ 
void direct_convolution( int C , int H , int W , int M , int K , int P, int S , int H_out , int W_out ,  int W_grid  , int tile_w  , float* X, float* W_filter, float* Y)
{
    int n , m , h , w , c , p , q ;
    n = blockIdx.x ;
    m = blockIdx.y ;
    h = (blockIdx.z / W_grid)*tile_w + threadIdx.y;
    w = (blockIdx.z % W_grid)*tile_w + threadIdx.x;

    H = H+P;
    W = W+P;

    if(h<H_out && w<W_out)
    {
        int temp=0;
        for( c = 0 ; c < C ; c++ )
        {
            for( p = 0 ; p < K ; p++ )
            {
                for( q = 0 ; q < K ; q++ )
                {
                    temp = temp + X[ n*(C*H*W) + c*(H*W) + (h*S+p)*(W) + (w*S+q)] * W_filter[ m*(C*K*K) + c*(K*K) + p*(K) + q] ;
                }
            }
        }
    
        Y[n*(M*H_out*W_out) + m*(H_out*W_out) + h*(W_out) + w] = temp;

    }


    }


int main(void)
{

  hipError_t err = hipSuccess;
 
  int N,C,M,K,H,W,P,S;
  printf("Enter the batch size : ");
  scanf("%d",&N);
  printf("\nEnter the channel size : ");
  scanf("%d",&C);
  printf("\nEnter the height of input feature maps : ");
  scanf("%d",&H);
  printf("\nEnter the width of input feature maps : ");
  scanf("%d",&W);
  printf("\nEnter the number of output feature maps : ");
  scanf("%d",&M);
  printf("\nEnter the kernel width(same as height) :  ");
  scanf("%d",&K);
  printf("\nEnter the padding size : ");
  scanf("%d",&P);
  printf("\nEnter the striding size : ");
  scanf("%d",&S);
 
  int size_input_matrix_0 = N * C * H * W * sizeof(float) ;         // size of input matrix before padding
 
  int size_input_matrix = N * C * (H+P) * (W+P) * sizeof(float) ;   // size of input matrix after padding
 
  int size_filter_matrix = M * C * K * K * sizeof(float) ; 
 
  int H_out = (H - K + P + S )/S;
  int W_out = (W - K + P + S )/S;
  int size_output_matrix = N * M * H_out * W_out * sizeof(float) ;
  
  float *X   = (float*)malloc(size_input_matrix_0 );     // X is input data matrix
  float *h_X = (float*)calloc(size_input_matrix/sizeof(float) , sizeof(float) );  // h_X is input data matrix after padding, this is sent to device for computation
  float *h_Y = (float*)malloc(size_output_matrix );                 // h_Y is the output data matrix
  float *h_W = (float*)malloc(size_filter_matrix );                 // h_W is the filter weights matrix
 
  if (h_X == NULL || h_Y == NULL || h_W == NULL || X == NULL )
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

  int n,c,m,h,w;
         
              #  /*  reading the input data matrix from user */
 
  printf("\n Enter the input matrix : \n");
  for(n = 0 ; n < N ; n++ )
  {
      for(c = 0 ; c < C ; c++ )
      {
          for( h = 0 ; h < H ; h++ )
          {
              for( w = 0 ; w < W ; w++)
              {
                  scanf("%f",&X[ n*(C*H*W) + c*(H*W) + h*(W) + w] );
              }
          }
      }
  }
 
                # /* padding the input data matrix with zeroes */
 
 for(n = 0 ; n < N ; n++ )
  {
      for(c = 0 ; c < C ; c++ )
      {
          for( h = P/2 ; h < H+P/2 ; h++ )
          {
              for( w = P/2 ; w < W+P/2 ; w++)
              {
                  h_X[ n*(C*(H+P)*(W+P)) + c*((H+P)*(W+P)) + h*(W+P) + w] =  X[ n*(C*H*W) + c*(H*W) + (h-P/2)*(W) + (w-P/2)];
              }
          }
      }
  }


                # /* printing the input data matrix after padding */

 for(n = 0 ; n < N ; n++ )
  {
      printf("n = %d\n",n);
      for(c = 0 ; c < C ; c++ )
      {
          printf(" channel - %d\n",c);
          for( h = 0 ; h < H+P ; h++ )
          {
              for( w = 0 ; w < W+P ; w++)
              {
                  printf("%f ",h_X[ n*(C*(H+P)*(W+P)) + c*((H+P)*(W+P)) + h*(W+P) + w] );
              }
           printf("\n");
          }
      }
  }
 
                # /* reading the filter weights matrix from user */
 
  printf("Enter the filter matrix : \n");
  for(m = 0 ; m < M ; m++ )
  {
      for(c = 0 ; c < C ; c++ )
      {
          for( h = 0 ; h < K ; h++ )
          {
              for( w = 0 ; w < K ; w++)
              {
                  scanf("%f",&h_W[ m*(C*K*K) + c*(K*K) + h*(K) + w] );
              }
          }
      }
  }
  

  float *d_X, *d_Y, *d_W;  // device vectors : d_X - input matrix , d_Y - output matrix , d_W - filter weights matix
 
                  # /* copying h_X to device */
 
  err = hipMalloc((void**)&d_X, size_input_matrix );
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to allocate device vector d_X (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err = hipMemcpy( d_X , h_X , size_input_matrix , hipMemcpyHostToDevice );
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to copy vector h_X from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
 
                    #  /* copying h_W to device */
 
  err = hipMalloc((void**)&d_W, size_filter_matrix );
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to allocate device vector d_W (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  err = hipMemcpy( d_W , h_W , size_filter_matrix, hipMemcpyHostToDevice );
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to copy vector h_W from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
 
                      # /* allocating memory for d_Y */
 
  err = hipMalloc((void**)&d_Y, size_output_matrix );
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to allocate device vector d_Y (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
    

  int tile_width = 2 , tile_height = 2;   
  int w_grid = ceil((W_out*1.0) / tile_width) ;
  int h_grid = ceil((H_out*1.0) / tile_height) ;
  
 
  int temp  = w_grid * h_grid;
  dim3 grid( N , M , temp );
  dim3 block( tile_width , tile_height , 1 );
 

          # /* calling the direct_convolution kernel */  
  direct_convolution<<< grid, block >>>( C, H , W , M , K , P , S ,  H_out , W_out , w_grid ,  tile_width ,  d_X , d_W , d_Y) ;



  err = hipGetLastError();

  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to launch reduce1 kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
 
  err = hipMemcpy(h_Y, d_Y, size_output_matrix , hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
 
              # /* printing the output matrix */
  for(n = 0 ; n < N ; n++ )
  {
      printf("N : %d \n",n);
      for(m = 0 ; m < M ; m++ )
      {
          printf("filter no. : %d \n", m);
          for( h = 0 ; h < H_out ; h++ )
          {
              for( w = 0 ; w < W_out ; w++)
              {
                  printf("%f ",h_Y[ n*(M * H_out * W_out) + m*(H_out * W_out) + h*(W_out) + w] );
              }
            printf("\n");
          }
      }
  }
 
          #  /* releasing all the device and host vectors */
 
  err = hipFree(d_X);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector X (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }      
  err = hipFree(d_Y);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector Y (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipFree(d_W);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to free device vector W (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }  
 
  free(h_X);
  free(h_W);
  free(h_Y);      

  
  return 0;
}