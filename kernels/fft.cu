#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>

//multiplication of 2 cufftComplex elements
//// One complex product for each thread scaled by total elements
__global__ void pointwise_product(hipfftComplex* d_outA, hipfftComplex* d_outB, float size, float scale)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;

    if (i < size)
    {
        float a,b;
        a = d_outA[i].x * d_outB[i].x - d_outA[i].y * d_outB[i].y;
        b = d_outA[i].x * d_outB[i].y + d_outA[i].y * d_outB[i].x;
        d_outA[i].x = a * scale;
        d_outA[i].y = b * scale;
    }
}

//flip filter about the center element
// <<H,W,D>>
__global__ void flip_filer(float* f_in, float* f_out, int H, int W, int D)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int dep = blockIdx.z*blockDim.z+threadIdx.z;
    int i = dep * H * W + col * W + row ;
 
    int new_col = H - col -1;
    int new_row = W - row - 1;
    int new_dep = D - dep - 1;
 
    int j = new_dep * H * W + new_col * W + new_row;

    if(col < H && row < W && dep < D)
    {
        f_out[j] = f_in[i];
    }
}

//Central element of the old_filter in the (0,0,0) position of the new_filter.
//(x,y,z) -> ((x-X/2)%X, (y-Y/2)%Y, (z-Z/2)%Z)
//new_filter[RHS] = old_filter[LHS]
// <<H,W,D>>
__global__ void align_filer(float* f_in, float* f_out, int H, int W, int D)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int dep = blockIdx.z*blockDim.z+threadIdx.z;
    int i = dep * H * W + col * W + row;
 
    int new_col = ((col - H/2) % H);
    int new_row = ((row - W/2) % W);
    int new_dep = ((dep - D/2) % D);
 
    new_col = new_col < 0 ? H + new_col: new_col;
    new_row = new_row < 0 ? W + new_row: new_row;
    new_dep = new_dep < 0 ? D + new_dep: new_dep;

    int j = new_dep * H * W + new_col * W + new_row;

    if(col < H && row < W && dep < D)
    {
        f_out[j] = f_in[i];
    }
}

//This utility function can be used to pad the filter(as it needs to be of the size of the input)
// or to pad the input when required
//<H+2*pad, W+2*pad,D>
__global__ void pad_input(float* f_in, float* f_out, int H, int W, int D, int pad)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int dep = blockIdx.z*blockDim.z+threadIdx.z;

    int new_H = H+2*pad; int new_W = W+2*pad; 
 
    int i = dep * new_H * new_W + col * new_W + row;
    int j = dep * H * W + (col - pad) *W+ (row - pad) ;

    if(col < new_H && row < new_W && dep < D)
    {
        if((col < pad || col > H+pad-1) || (row < pad || row > W+pad-1)) f_out[i] = 0;
        else f_out[i] = f_in[j];
    }
}

//crop and output the required size output (O = ((W - k + 2 * P)) + 1)
//<<H, W, D>>
__global__ void crop(float* f_in, float* f_out, int H, int W, int O_H, int O_W, int D)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    //int dep = blockIdx.z*blockDim.z+threadIdx.z;
    int i =  (D/2) * H * W + col * W + row;
    
    int crop_H = (H - O_H)/2;
    int crop_W = (W - O_W)/2;

    int j = (col - crop_H) * O_W+ (row - crop_W);

    if(col < H && row < W)
    {
        if(col >= crop_H && col < H - crop_H && row >= crop_W && row < W - crop_W)f_out[j] = f_in[i];
    }
}

//stride and output the required size output (O = ((W - k + 2 * P)/stride) + 1)
//call this only if stride is not 1
//<<H, W, 1>>
__global__ void stride_(float* f_in, float* f_out, int H, int W, int stride)
{
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    //int dep = blockIdx.z*blockDim.z+threadIdx.z;
 
    int O_H = H/stride + 1;
    int O_W = W/stride + 1;
    
    int i = col * W + row; 
    //+ dep * H * W;

    if(col < H && row < W && (col%stride == 0) && (row%stride == 0))
    {
        int j = (col/stride) * O_W + (row/stride) ;
        // dep * H * W;
        f_out[j] = f_in[i];
    }
}

float* conv_operation(float* filter_align, float* input_layer_pad, int H, int W, int D, int BS)
{
    int N[3] = {D,H, W};
    hipfftReal* d_inA, *d_inB;
    hipfftComplex* d_outA, *d_outB;
    hipfftHandle fwplanA, fwplanB, bwplan;
    size_t real_size = BS * D* W * H * sizeof(hipfftReal);
    size_t complex_size = BS * D * W * (H/2 + 1) * sizeof(hipfftComplex);

    hipMalloc((void**)&d_inA, real_size);
    hipMalloc((void**)&d_inB, real_size);
    hipMalloc((void**)&d_outA, complex_size);
    hipMalloc((void**)&d_outB, complex_size);
    hipMemset(d_inA,0,real_size);
    hipMemset(d_inB,0,real_size);
 
    float * filter_align_in = (float *)malloc(real_size);
    for(int  i = 0; i < BS; i++)
    {
        hipMemcpy(&filter_align_in[i * D * H * W], filter_align, real_size/BS,  hipMemcpyHostToHost);
    }
 
    hipMemcpy(d_inA, filter_align_in, real_size,  hipMemcpyHostToDevice);
    hipMemcpy(d_inB, input_layer_pad, real_size, hipMemcpyHostToDevice); //update inpute_layer
  
    //cufftPlan3d(&fwplanA, D, H, W, CUFFT_R2C);
    //cufftPlan3d(&fwplanB, D, H, W, CUFFT_R2C);
    //cufftPlan3d(&bwplan, D, H, W, CUFFT_C2R);

    
    hipfftPlanMany(&fwplanA, 3, N, NULL, 0,0,NULL,0,0, HIPFFT_R2C ,BS);
    hipfftPlanMany(&fwplanB, 3, N, NULL, 0,0,NULL,0,0, HIPFFT_R2C ,BS);
    hipfftPlanMany(&bwplan, 3, N, NULL, 0,0,NULL,0,0, HIPFFT_C2R ,BS);
    
    hipfftExecR2C(fwplanA, d_inA, d_outA);
    hipfftExecR2C(fwplanB, d_inB, d_outB);

    int blocksx = ceil((BS * D*W*(H/2 + 1)) / 256.0f);
    dim3 threads4(256);
    dim3 grid4(blocksx);
    pointwise_product<<<grid4, threads4>>>(d_outA, d_outB, (BS * D*W*(H/2 + 1)), 1.0f/(H*W*D));

    hipfftExecC2R(bwplan, d_outA, d_inA);
    
    float* result1 = new float[BS * D*W*2*((H/2 + 1)) ];
    hipMemcpy(result1, d_inA, real_size,hipMemcpyDeviceToHost);
    return result1;
}

// input arguments are input_layer, kernel, padding, stride, batch_size, input_layer dimensions, kernel dimensions
// Operations: pad(input), pad(filter), align(filter), output = convolve(input,filter), crop(output), stride(output)  

float* convolve_FFT(float * input_layer, float * kernel, int pad, int stride, int batch_size, int* il_dim, int* kernel_dim)
{
  //////initializations
  int H = il_dim[0], W = il_dim[1], D = il_dim[2]; int BS = batch_size;
  int fH = kernel_dim[0], fW = kernel_dim[1] , fD = kernel_dim[2];
  hipError_t err = hipSuccess;
  //////////////////////////

  ///////pad input
  int new_H = H+2*pad; int new_W = W+2*pad;
  float *input_layer_pad = (float *)malloc(BS * D * new_H* new_W *sizeof(float));
  for(int i = 0; i < BS; i++)
  {
      float *pad_input_in = NULL; hipMalloc((void **)&pad_input_in, H * W * D * sizeof(float));
      float *pad_input_out = NULL; hipMalloc((void **)&pad_input_out, new_H * new_W * D * sizeof(float));
      hipMemcpy(pad_input_in, &input_layer[i * D * H * W], H * W * D * sizeof(float) , hipMemcpyHostToDevice);
      
      dim3 threads1(1,1,1);
      dim3 grid1(new_H,new_W,D);
      pad_input<<<grid1,threads1>>>(pad_input_in, pad_input_out, H,W,D,pad);
      err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch pad input (error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
      
      hipMemcpy(&input_layer_pad[i * D * new_H * new_W], pad_input_out , new_H * new_W * D * sizeof(float), hipMemcpyDeviceToHost);
      hipFree(pad_input_in); hipFree(pad_input_out);
  }
  H = new_H; W = new_W;
  //////pad input end

  //////flip filter
  float *filter_flip = (float *)malloc(fH * fW * fD *sizeof(float));
  float *f_A = NULL; hipMalloc((void **)&f_A, fH * fW * fD * sizeof(float));
  float *f_B = NULL; hipMalloc((void **)&f_B, fH * fW * fD * sizeof(float));
  hipMemcpy(f_A, kernel , fH * fW * fD * sizeof(float), hipMemcpyHostToDevice);

  dim3 threads0(1,1,1);
  dim3 grid0(fH,fW,fD);
  flip_filer<<<grid0, threads0>>>(f_A, f_B, fH,fW,fD);
  err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch align_filter(error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
  
  hipMemcpy(filter_flip, f_B, fH*fW*fD*sizeof(float), hipMemcpyDeviceToHost);

 ///////flip filter end

  ///////pad filter 
  int fpad = (new_H - fH)/2; 
  int new_fH = fH+2*fpad; int new_fW = fW+2*fpad;
  float *pad_filter_in = NULL; hipMalloc((void **)&pad_filter_in, fH * fW * fD * sizeof(float));
  float *pad_filter_out = NULL; hipMalloc((void **)&pad_filter_out, new_fH * new_fW * D * sizeof(float));
  float *filter_pad = (float *)malloc(new_fH* new_fW * D *sizeof(float));

  hipMemcpy(pad_filter_in, filter_flip , fH * fW * fD * sizeof(float) , hipMemcpyHostToDevice);
  dim3 threads2(1,1,1);
  dim3 grid2(new_fH,new_fW,D);

  pad_input<<<grid2,threads2>>>(pad_filter_in, pad_filter_out, fH,fW,D,fpad);
  err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch pad filter(error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
  hipMemcpy(filter_pad, pad_filter_out , new_fH * new_fW * D * sizeof(float), hipMemcpyDeviceToHost);
  fH = new_fH; fW = new_fW;
  //////pad filter end
 
  ///////align filter begin
  float *filter_align = (float *)malloc(fH * fW * fD *sizeof(float));
  float *d_A = NULL; hipMalloc((void **)&d_A, fH * fW * fD * sizeof(float));
  float *d_B = NULL; hipMalloc((void **)&d_B, fH * fW * fD * sizeof(float));
  hipMemcpy(d_A, filter_pad, fH * fW * fD * sizeof(float), hipMemcpyHostToDevice);

  dim3 threads3(1,1,1);
  dim3 grid3(fH,fW,fD);
  align_filer<<<grid3, threads3>>>(d_A, d_B, fH,fW,fD);
  err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch align_filter(error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
  
  hipMemcpy(filter_align, d_B, fH*fW*fD*sizeof(float), hipMemcpyDeviceToHost);
  ///////align filter end
 
  ///////Convolve begin (FFT, Pointwise prodcut, IFFT)
  float* conv_result = conv_operation( filter_align, input_layer_pad, H, W, D, BS);
  //////convolve end

  ////////crop output
  fH = kernel_dim[0]; fW = kernel_dim[1] ; fD = kernel_dim[2];
  int oH = H - fH + 1; int oW = W - fW + 1;
  float* result2 = (float*)malloc(BS * oW*oH* sizeof(float));
  for(int i = 0; i < BS; i++)
  {
    float *crop_out = NULL; hipMalloc((void **)&crop_out, oH * oW * sizeof(float));
    float *crop_in = NULL; hipMalloc((void **)&crop_in, D * H * W * sizeof(float));
    hipMemcpy(crop_in, &conv_result[i * D * H* W],  D * H * W * sizeof(float),hipMemcpyHostToDevice);
    
    dim3 threads5(1,1,1);
    dim3 grid5(H,W,1);
    crop<<<grid5, threads5>>>(crop_in, crop_out, H, W, oH, oW, D);
    err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch crop(error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
    
    hipMemcpy(&result2[i*oW*oH], crop_out, oW*oH* sizeof(float) ,hipMemcpyDeviceToHost);
  }
  
  ///////crop output end

  ///////stride output stride_(float* f_in, float* f_out, int H, int W, int stride)
  if(stride != 1)
  {
      int sH = oH / stride + 1; int sW = oW / stride + 1; 
      float* result_s = (float *)malloc(BS* sH*sW*sizeof(float));
   
      for(int i = 0; i < BS ; i++)
      {
          float *stride_in = NULL; hipMalloc((void **)&stride_in, oH * oW * sizeof(float));
          float *stride_out = NULL; hipMalloc((void **)&stride_out, sH * sW * sizeof(float));
          hipMemcpy(stride_in, &result2[i * oW* oH], oW*oH* sizeof(float) ,hipMemcpyHostToDevice);
          dim3 threads6(1,1,1);
          dim3 grid6(oH,oW,1);
          stride_<<<grid6, threads6>>>(stride_in, stride_out ,oH, oW, stride);
          err = hipGetLastError(); if(err!=hipSuccess){fprintf(stderr, "Failed to launch stride(error code %s)!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);}
          hipMemcpy(&result_s[i*sH*sW], stride_out , sH * sW * sizeof(float) ,hipMemcpyDeviceToHost);
      }
      result2 = result_s;
  }
  ///////stride output end
  
  return result2;
  
}