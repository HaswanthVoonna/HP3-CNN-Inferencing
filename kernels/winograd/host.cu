#include "header.h"
#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)

using namespace std;

void gpu_error(hipError_t const &code) 
{
    cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << endl;
    exit( EXIT_FAILURE );
}

// forward(int out_channels, int input_channels, int kernel_height, int kernel_width, int padding, int stride, float* kernel_weights, int batchsize_of_data, int input_height, int input_width, float* input)

void rearrange(int ch, int bs, int h, int w, float *& in)
{
    // ch   : input_channels
    // bs   : batchsize_of_data
    // h    : input_height
    // w    : input_width
    // in   : reference to float pointer input
    // This function transforms the input from bs x ch x h x w to ch x h x w x bs

    float *newin = new float[ch * h * w * bs];
    float *newiter = newin;
    int ch_h_w = ch*h*w, h_w = h*w; 
    LOOP(ch)
    {
        LOOP(h)
        {
            LOOP(w)
            {
                LOOP(bs)
                {
                    *(newiter++) = in[tbs * ch_h_w + tch * h_w + th * w + tw];
                }
            }
        }
    }

    delete in;

    in = newin;

}

int main(void) {
    auto engine = default_random_engine(time(nullptr));
    auto rng = uniform_real_distribution<float>();

    size_t n; cin >> n;
    if( n > N_MAX ) {
        cerr << "Size " << n << " too large." << endl;
        return EXIT_FAILURE;
    }

    size_t mat_size = n * n;
    float P[mat_size], Q[mat_size], *d_P = nullptr;
    for(size_t i=0; i<mat_size; i++) Q[i] = P[i] = rng(engine);

    hipError_t code = hipSuccess;
    if( (code = hipMalloc((void**)&d_P, sizeof P)) != hipSuccess ) gpu_error(code);
    if( (code = hipMemcpy(d_P, P, sizeof P, hipMemcpyHostToDevice)) != hipSuccess ) gpu_error(code);

    solve<<< 1,dim3(n,n) >>> (d_P, n);

    if( (code = hipMemcpy(Q, d_P, sizeof P, hipMemcpyDeviceToHost)) != hipSuccess ) gpu_error(code);
    if( (code = hipFree(d_P)) != hipSuccess ) gpu_error(code);
    d_P = nullptr;

    if( not check(n, P, Q) ) {
        cerr << "Test failed." << endl;
        return EXIT_FAILURE;
    }

    cout << "Test passed." << endl;
    return EXIT_SUCCESS;
}
