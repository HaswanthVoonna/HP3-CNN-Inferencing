#include "hip/hip_runtime.h"
%%cuda --name helloCUDA.cu
#include <iostream>
#include <random>
#include <algorithm>

#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)

using namespace std;

void gpu_error(hipError_t const &code) 
{
    if(code != hipSuccess)
    {
        cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << endl;
        exit( EXIT_FAILURE );
    }
}

__global__ void tile(float *devin, float *devout, float *devsum, int h, int w)
{
    float thrtile[4][4];
    
    int /*bs,*/ p, q, ch;
    // bs = gridDim.x;
    p = gridDim.y;
    q = gridDim.z;
    ch = blockDim.x;
    
    int tbs, tp, tq, tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    tch = threadIdx.x;

    // copy the tiles to thrtile

    int offset1 = (tbs*ch + tch)*h*w;

    // float *t = thrtile;
 
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
    {
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
        {
            thrtile[i][j] = devin[offset1 + th*w + tw];
        }
    }

    // copy thrtile to devout for testing

    int offset2 = (((tbs*p + tp)*q + tq)*ch + tch)*16;

    for(int i = 0; i < 4; i++)
    {
        for(int j = 0; j < 4; j++)
        {
            devout[offset2 + i*4 + j] = thrtile[i][j];
        }
    }

    // sum along the channels, using log n summing

    // int k = ch, j = tch;

    int offset3 = ((tbs*p + tp)*q + tq)*ch*16;

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            for(int i = 0; i < 4; i++)
            {
                for(int j = 0; j < 4; j++)
                {
                    devout[offset3 + tch*16 + i*4 + j] += devout[offset3 + (tch+s)*16 + i*4 + j];
                }
            }
        }
        __syncthreads();
    }

    if(tch/*%ch*/ == 0) // can do with tch == 0
    {
        int offset = ((tbs*p + tp)*q + tq)*16;
        for(int i = 0; i < 4; i++)
        {
            for(int j = 0; j < 4; j++)
            {
                devsum[offset + i*4 + j] = devout[offset3 + /*tch*16*/ +i*4 + j];
            }
        }
    }

}

__global__ void paddev(float *devin, float *devinnopad, int h, int w, int pad)
{
    int newh = gridDim.y;
    int neww = gridDim.z;
    int tbsch = blockIdx.x;
    int tnewh = blockIdx.y;
    int tneww = blockIdx.z;
    int newhw = newh*neww;
    int hw = h*w;
    int th = tnewh-pad;
    int tw = tneww-pad;
    
    if(th >= 0 && th < h && tw >= 0 && tw < w)
    {
        devin[tbsch*newhw + tnewh*neww + tneww] = devinnopad[tbsch*hw + th*w + tw];
    }
    else
    {
        devin[tbsch*newhw + tnewh*neww + tneww] = 0;
    }
    
}

void tilehost(int och, int ch, int bs, int &h, int &w, float *&in, int &p, int &q, int &outsize, float *&out, int &sumsize, float *&sum, int pad, float *&padded)
{
    float *devin, *devinnopad;
    int insize = bs * ch * h * w * sizeof(float);
    gpu_error(hipMalloc((void **) & devinnopad, insize));
    gpu_error(hipMemcpy(devinnopad, in, insize, hipMemcpyHostToDevice));

    int newh, neww;
    newh = h + 2*pad;
    neww = w + 2*pad;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    insize = bs * ch * newh * neww * sizeof(float);
    gpu_error(hipMalloc((void **) & devin, insize));

    // call padding
    dim3 padgrid(bs*ch, newh, neww);
    dim3 padblock(1, 1, 1);
 
    paddev<<<padgrid,padblock>>>(devin, devinnopad, h, w, pad);

    gpu_error(hipFree(devinnopad));
 
    padded = new float[insize/sizeof(float)];
    
    gpu_error(hipMemcpy(padded, devin, insize, hipMemcpyDeviceToHost));
    
    h = newh;
    w = neww;

    
    // int p, q;
    p = max((h-2)/2, 0);
    q = max((w-2)/2, 0);
    
    float *devout, *devsum;
    devout = devsum = nullptr;
    outsize = bs * p * q * ch * 4 * 4 * sizeof(float);
    sumsize = bs * p * q * 4 * 4 * sizeof(float);

    gpu_error(hipMalloc((void **) & devout, outsize));
    gpu_error(hipMalloc((void **) & devsum, sumsize));
    
    // call the kernel function for tiling
    
    dim3 grid(bs, p, q);  // 3-D
    dim3 block(ch, 1, 1); // 1-D

    tile<<<grid, block>>>(devin, devout, devsum, h, w);

    // copy from device to host to out.

    delete in;
    out = new float[outsize/sizeof(float)];
    sum = new float[sumsize/sizeof(float)];

    gpu_error(hipMemcpy(out, devout, outsize, hipMemcpyDeviceToHost));
    gpu_error(hipMemcpy(sum, devsum, sumsize, hipMemcpyDeviceToHost));

    gpu_error(hipFree(devin));
    gpu_error(hipFree(devout));
    gpu_error(hipFree(devsum));
    
}

void padding(float *&in, int bs, int ch, int &h, int &w, int pad)
{
    // Here, after adding pad we also round up h, w to become a multiple of tile.
    // This is done such that the actual matrix is present at top left of this matrix.

    int newh, neww;
    newh = h + 2*pad;
    neww = w + 2*pad;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    int slices = bs*ch;
    int newhw = newh*neww;
    float *newin = new float[slices*newhw];
    float *tin = in, *tnewin = newin;
    LOOP(slices)
    {
        LOOP(newh)
        {
            LOOP(neww)
            {
                if(tnewh >= pad && tnewh-pad < h && tneww >= pad && tneww-pad < w)
                {
                    *(tnewin++) = *(tin++);
                }
                else
                {
                    *(tnewin++) = 0;
                }
            }
        }
    }

    delete in;
    in = newin;

    h = newh;
    w = neww;

}

int main(void) 
{
    auto engine = default_random_engine(time(nullptr));
    auto rng = uniform_real_distribution<float>();

    int bs, ch, h, w, p, q, oldh, oldw, pad;
    
    bs = 3;
    ch = 2;
    oldh = h = 3;
    oldw = w = 3;
    pad = 1;
    
    int insize = bs * ch * h * w * sizeof(float);
    int outsize, sumsize;
 
    float *in = new float[insize/sizeof(float)];
    float *t = in;
    float *out, *sum, *padded;
 
    LOOP(bs)
    {
        LOOP(ch)
        {
            LOOP(h)
            {
                LOOP(w)
                {
                    *(t++) = rng(engine);
                }
            }
        }
    }
 
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(ch)
        {
            cout<<"{ ";
            LOOP(h)
            {
                cout<<"{ ";
                LOOP(w)
                {
                    cout<<in[((tbs*ch+tch)*h+th)*w+tw]<<" ";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }

    //cout<<"\nPadding\n";

    //padding(in, bs, ch, h, w, pad);

   // cout<<"\nPadding done\n";
    
    /*
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(ch)
        {
            cout<<"{ ";
            LOOP(h)
            {
                cout<<"{ ";
                LOOP(w)
                {
                    cout<<in[((tbs*ch+tch)*h+th)*w+tw]<<" ";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
    */

    cout<<"\nPadding and Tiling and Summing\n";

    tilehost(1, ch, bs, h, w, in, p, q, outsize, out, sumsize, sum, pad, padded);
 
    cout<<"\nPadding finished\n\n";
 
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(ch)
        {
            cout<<"{ ";
            LOOP(h)
            {
                cout<<"{ ";
                LOOP(w)
                {
                    cout<<padded[((tbs*ch+tch)*h+th)*w+tw]<<" ";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
    
    cout<<"\nTiling finished\n\n";

    /*
    
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(p)
        {
            cout<<"{ ";
            LOOP(q)
            {
                cout<<"{ ";
                LOOP(ch)
                {
                    cout<<"{ ";
                    for(int i = 0; i < 4; i++)
                    {
                        for(int j = 0; j < 4; j++)
                        {
                            cout<<out[((((tbs*p+tp)*q+tq)*ch+tch)*4+i)*4+j]<<",";
                        }
                        cout<<";\n";
                    }
                    cout<<"}\n";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
 
    */

    cout<<"\nSumming finished\n\n";

    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(p)
        {
            cout<<"{ ";
            LOOP(q)
            {
                cout<<"{ ";
                for(int i = 0; i < 4; i++)
                {
                    for(int j = 0; j < 4; j++)
                    {
                        cout<<sum[(((tbs*p+tp)*q+tq)*4+i)*4+j]<<",";
                    }
                    cout<<";\n";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }

    return 0;
}