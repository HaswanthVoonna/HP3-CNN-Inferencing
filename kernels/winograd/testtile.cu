#include "hip/hip_runtime.h"
%%cuda --name helloCUDA.cu
#include <iostream>
#include <random>
#include <algorithm>

#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)

using namespace std;

__global__ precompute(int out_channels, int input_channels, float* kernel_weights, float *U)
{
    int x = threadIdx.x;
    int y = blockDim.x;;
    int bid = blockIdx.x;
    int offset = bid*y + x;
    int m = 2, n = 3;
    
    float g[4][3] = {
        {1, 0, 0},
        {0.5, 0.5, 0.5},
        {0.5, -0.5, 0.5},
        {0, 0, 1}
    };
    
    float g_t[3][4] ={
        {1, 0.5, 0.5, 0},
        {0, 0.5, -0.5, 0},
        {0, 0.5, 0.5, 1}
    };

    float *temp = (float *)malloc(out_channels*input_channels*3*4*sizeof(float));
    for(int i = 0; i <3; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[offset*3*4+i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                temp[offset*3*4+i*4+j] += kernel_weights[offset*3*3+i*3+k] * g_t[k][j];
            }
        }
    }

    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            U[offset*4*4+i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                U[offset*4*4+i*4+j] += g[i][k] * temp[offset*3*4+k*4+j];
            }
        }
    }
}

__global__ void uv(int tch, int out_channels, float *fin, float *U, float V[4][4])
{
    int x = threadIdx.x;
    int offset = x*out_channels+tch;

    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            fin[x*out_channels*4*4 + i*4 + j] = U[offset*4*4+i*4+j]*V[i][j];
            
        }
    }
}

__global__ void tile(float *devin, float *devout, float *devsum, float *U, int h, int w, int och)
{
    float thrtile[4][4];
    
    int /*bs,*/ p, q, ch;
    // bs = gridDim.x;
    p = gridDim.y;
    q = gridDim.z;
    ch = blockDim.x;
    
    int tbs, tp, tq, tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    tch = threadIdx.x;

    // copy the tiles to thrtile

    int offset1 = (tbs*ch + tch)*h*w;

    // float *t = thrtile;
 
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
    {
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
        {
            thrtile[i][j] = devin[offset1 + th*w + tw];
        }
    }
    float B[4][4] = {
        {1,0,0,0},
        {0,1,-1,1},
        {-1,1,1,0},
        {0,0,0,-1}
    };

    float B_t[4][4] = {
        {1,0,-1,0},
        {0,1,1,0},
        {0,-1,1,0},
        {0,1,0,-1}
    };
    //Calculation of V
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += thrtile[i][k] * B[k][j];
            }
     
        }
    }

    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            V[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                V[i][j] += B_t[i][k] * temp[k][j];
            }
        }
    }
/

    float *fin = (float *)malloc(och*4*4);
    uv<<<1,och>>>(tch, och, fin, U, V); 

    // copy thrtile to devout for testing

    int offset2 = (((tbs*p + tp)*q + tq)*ch + tch)*16;
    LOOP(och)
    {
        for(int i = 0; i < 4; i++)
        {
            for(int j = 0; j < 4; j++)
            {
                devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] = fin[(toch*4+i)*4+j];
            }
        }
    }

    // sum along the channels, using log n summing

    // int k = ch, j = tch;

    int offset3 = ((tbs*p + tp)*q + tq)*ch*16;

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            LOOP(och)
            {
                for(int i = 0; i < 4; i++)
                {
                    for(int j = 0; j < 4; j++)
                    {
                        devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] += devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+(tch+s))*4 + i)*4 + j];
                    }
                }
            }
        }
        __syncthreads();
    }

    if(tch/*%ch*/ == 0) // can do with tch == 0
    {
        int offset = ((tbs*p + tp)*q + tq)*16;
        LOOP(och)
        {    
            for(int i = 0; i < 4; i++)
            {
                for(int j = 0; j < 4; j++)
                {
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch)*4 + i)*4 + j];
                }
            }
        }
    }

}

void gpu_error(hipError_t const &code) 
{
    if(code != hipSuccess)
    {
        cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << endl;
        exit( EXIT_FAILURE );
    }
}

void tilehost(int och, int ch, int bs, int h, int w, float *&in, int &p, int &q, int &outsize, float *&out, int &sumsize, float *&sum, float *kernel_weights)
{
    // int p, q;
    p = max((h-2)/2, 0);
    q = max((w-2)/2, 0);
    
    float *devin, *devout, *devsum;
    devin = devout = devsum = nullptr;
    int insize = bs * ch * h * w * sizeof(float);
    outsize = bs * och * p * q * ch * 4 * 4 * sizeof(float);
    sumsize = bs * och * p * q * 4 * 4 * sizeof(float);

    gpu_error(hipMalloc((void **) & devin, insize));
    gpu_error(hipMalloc((void **) & devout, outsize));
    gpu_error(hipMalloc((void **) & devsum, sumsize));
    
    gpu_error(hipMemcpy(devin, in, insize, hipMemcpyHostToDevice));

    // call the kernel function for tiling
    
    float *U = (float *)malloc(och*ch*4*4*sizeof(float));
    precompute<<<och, ch>>>(och, ch, kernel_weights, U);

    dim3 grid(bs, p, q);  // 3-D
    dim3 block(ch, 1, 1); // 1-D
    tile<<<grid, block>>>(devin, devout, devsum, U, h, w, och);

    // copy from device to host to out.

    delete in;
    out = new float[outsize/sizeof(float)];
    sum = new float[sumsize/sizeof(float)];

    gpu_error(hipMemcpy(out, devout, outsize, hipMemcpyDeviceToHost));
    gpu_error(hipMemcpy(sum, devsum, sumsize, hipMemcpyDeviceToHost));

    gpu_error(hipFree(devin));
    gpu_error(hipFree(devout));
    gpu_error(hipFree(devsum));
    
}

int main(void) 
{
    auto engine = default_random_engine(time(nullptr));
    auto rng = uniform_real_distribution<float>();

    int bs, ch, h, w, p, q;
    
    bs = 1;
    ch = 2;
    h = 9;
    w = 9;
    
    int insize = bs * ch * h * w * sizeof(float);
    int outsize, sumsize;
 
    float *in = new float[insize/sizeof(float)];
    float *t = in;
    float *out, *sum;
 
    LOOP(bs)
    {
        LOOP(ch)
        {
            LOOP(h)
            {
                LOOP(w)
                {
                    *(t++) = rng(engine);
                }
            }
        }
    }
 
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(ch)
        {
            cout<<"{ ";
            LOOP(h)
            {
                cout<<"{ ";
                LOOP(w)
                {
                    cout<<in[((tbs*ch+tch)*h+th)*w+tw]<<" ";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }

    cout<<"\nTiling and Summing\n";

    tilehost(1, ch, bs, h, w, in, p, q, outsize, out, sumsize, sum, kernel_weights);
    
    cout<<"\nTiling finished\n\n";

    /*
    
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(p)
        {
            cout<<"{ ";
            LOOP(q)
            {
                cout<<"{ ";
                LOOP(ch)
                {
                    cout<<"{ ";
                    for(int i = 0; i < 4; i++)
                    {
                        for(int j = 0; j < 4; j++)
                        {
                            cout<<out[((((tbs*p+tp)*q+tq)*ch+tch)*4+i)*4+j]<<",";
                        }
                        cout<<";\n";
                    }
                    cout<<"}\n";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
 
    */

    cout<<"\nSumming finished\n\n";

    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(p)
        {
            cout<<"{ ";
            LOOP(q)
            {
                cout<<"{ ";
                for(int i = 0; i < 4; i++)
                {
                    for(int j = 0; j < 4; j++)
                    {
                        cout<<sum[(((tbs*p+tp)*q+tq)*4+i)*4+j]<<",";
                    }
                    cout<<";\n";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }

    return 0;
}