#include "hip/hip_runtime.h"
//%%cuda --name helloCUDA.cu
#include <iostream>
#include <random>
#include <algorithm>
#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)
#define cudaSafeCall(call)  \
        do {\
            hipError_t err = call;\
            if (hipSuccess != err) \
            {\
                std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__ << "): " \
                    << hipGetErrorString(err);\
                exit(EXIT_FAILURE);\
            }\
        } while(0)

using namespace std;

void gpu_error(hipError_t const &code) {
    if(code != hipSuccess)
    {
        cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << endl;
        exit( EXIT_FAILURE );
    }
}

__global__ void precompute(int och, int ch, float* kernel_weights, float *U)
{
    int x = threadIdx.x;
    int bid = blockIdx.x;
    int offset = bid*ch + x;
   
    float g[4][3] = {
        {1, 0, 0},
        {0.5, 0.5, 0.5},
        {0.5, -0.5, 0.5},
        {0, 0, 1}
    };
    
    float g_t[3][4] ={
        {1, 0.5, 0.5, 0},
        {0, 0.5, -0.5, 0},
        {0, 0.5, 0.5, 1}
    };
    float *temp = (float *)malloc(3*4*sizeof(float));
    for(int i = 0; i <3; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                temp[i*4+j] += kernel_weights[offset*3*3+i*3+k] * g_t[k][j];
            }
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            U[offset*4*4+i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                U[offset*4*4+i*4+j] += g[i][k] * temp[k*4+j];
            }
        }
    }
}
__global__ void uv(int tch, int ch, float *devfin, float *U,  float *V)
{
    int x = threadIdx.x;
    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[x*4*4 + i*4 + j] = U[((x*ch+tch)*4+i)*4+j]*V[i*4+j];            
}
__global__ void amul(int tbs, int tp, int tq, int bs, int och, int p, int q, float *devsum, float *devY)
{
    float A_t[2][4] = {
        {1, 1, 1, 0},
        {0, 1, -1,-1}
    };
    float A[4][2] = {
        {1,0},
        {1,1},
        {1,-1},
        {0,1}
    };
    int x = threadIdx.x;
    float *temp = (float *)malloc(2*4*sizeof(float));
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i*4+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i*4+j] += A_t[i][k] * devsum[((((tbs*och+x)*p+tp)*q+tq)*4+k)*4+j];
            }
        }
    }
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <2; ++j)
        {
            devY[((((tbs*och+x)*p+tp)*q+tq)*2+i)*2+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                devY[((((tbs*och+x)*p+tp)*q+tq)*2+i)*2+j] += temp[i*4+k] * A[k][j];
            }
        }
    }
}

__global__ void paddev(float *devin, float *devinnopad, int h, int w, int pad)
{
    int newh = gridDim.y;
    int neww = gridDim.z;
    int tbsch = blockIdx.x;
    int tnewh = blockIdx.y;
    int tneww = blockIdx.z;
    int newhw = newh*neww;
    int hw = h*w;
    int th = tnewh-pad;
    int tw = tneww-pad;
    
    if(th >= 0 && th < h && tw >= 0 && tw < w)
        devin[tbsch*newhw + tnewh*neww + tneww] = devinnopad[tbsch*hw + th*w + tw];
    else
        devin[tbsch*newhw + tnewh*neww + tneww] = 0;
    
}

__global__ void cutpad(float  *devY, float *devcutY, int oph,int opw)
{
    int p = gridDim.y;
    int q = gridDim.z;
    int tbsch = blockIdx.x;
    int tp = blockIdx.y;
    int tq = blockIdx.z;
    //int newhw = newh*neww;
    //int pq4 = p*q*4;
    int ophopw = oph*opw;
    for(int i = 0; i < 2; i++)
    {
        for(int j = 0; j < 2; j++)
        {
            if(tp*2 + i < oph && tq*2 + j < opw)
                devcutY[tbsch*ophopw + (tp*2+i)*opw + (tq*2+j)] = devY[(((tbsch*p + tp)*q +tq)*2 + i)*2  + j];
        }
    }
}
    
__global__ void tile(int bs, int p, int q, int ch, float *devin, float *devout, float *devsum, float *devY, float *devU, int h, int w, int och, float *&devfin)
{
    float thrtile[4][4];    
    int tbs, tp, tq, tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    tch = threadIdx.x;
    // copy the tiles to thrtile
    int offset1 = (tbs*ch + tch)*h*w;
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
            thrtile[i][j] = devin[offset1 + th*w + tw];

    float B[4][4] = {
        {1,0,0,0},
        {0,1,-1,1},
        {-1,1,1,0},
        {0,0,0,-1}
    };
    float B_t[4][4] = {
        {1,0,-1,0},
        {0,1,1,0},
        {0,-1,1,0},
        {0,1,0,-1}
    };
    //Calculation of V
    float temp[4][4];
    float *V = (float *)  malloc(16*sizeof(float));
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += thrtile[i][k] * B[k][j];
            }   
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            V[i*4+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                V[i*4+j] += B_t[i][k] * temp[k][j];
            }
        }
    }
    float *fin = (float *)malloc(och*4*4*sizeof(float));
    uv<<<1,och>>>(tch, ch, fin, devU, V); 
    hipDeviceSynchronize();

    for(int toch = 0; toch<och; toch++)
        for(int i = 0; i < 4; i++)
            for(int j = 0; j < 4; j++)
               devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] = fin[(toch*4+i)*4+j];

    // sum along the channels, using log n summing

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] += devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+(tch+s))*4 + i)*4 + j];
        }
        __syncthreads();
    }
    if(tch == 0) 
    {
        LOOP(och)
            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch)*4 + i)*4 + j];
    }
  if(tch == 0)
  {
      amul<<<1,och>>>(tbs, tp, tq, bs, och, p, q, devsum, devY);
      hipDeviceSynchronize();
  }
}
void tilehost(int och, int ch, int bs, int h, int w, int pad, float *&in, int &p, int &q, int &oph, int &opw, int &outsize, float *&out, int &sumsize, float *&sum, int &ysize, float *&Y, float *&cutY, float *kwt)
{
    float *devin, *devinnopad;
    int insize = bs * ch * h * w * sizeof(float);
    int newh, neww;
 
    gpu_error(hipMalloc((void **) & devinnopad, insize));
    gpu_error(hipMemcpy(devinnopad, in, insize, hipMemcpyHostToDevice));

    newh = h + 2*pad;
    neww = w + 2*pad;
    oph = newh-2;
    opw = neww-2;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    insize = bs * ch * newh * neww * sizeof(float);
    gpu_error(hipMalloc((void **) & devin, insize));

    // call padding
    dim3 padgrid(bs*ch, newh, neww);
    dim3 padblock(1, 1, 1);
 
    paddev<<<padgrid,padblock>>>(devin, devinnopad, h, w, pad);

    gpu_error(hipFree(devinnopad));
    h = newh;
    w = neww;
    p = max((h-2)/2, 0);
    q = max((w-2)/2, 0);
    
    float *devout, *devsum, *devkwt, *devU, *devY, *devcutY;
    float *devfin;
    devout = devsum = nullptr;
 
    int kwtsize = och*ch*3*3*sizeof(float);
    int finsize = bs * p * q * ch * och * 4 * 4 * sizeof(float);
    outsize = bs * och * p * q * ch * 4 * 4 * sizeof(float);
    sumsize = bs * och * p * q * 4 * 4 * sizeof(float);
    ysize = bs * och * p * q * 2 * 2 * sizeof(float);
    int usize = och*ch*4*4*sizeof(float);
    int cutsize = bs*och*oph*opw*sizeof(float);
 
    gpu_error(hipMalloc((void **) & devout, outsize));
    gpu_error(hipMalloc((void **) & devsum, sumsize));
    gpu_error(hipMalloc((void **) & devkwt, kwtsize));
    gpu_error(hipMalloc((void **) & devU, usize));
    gpu_error(hipMalloc((void **) & devfin, finsize));
    gpu_error(hipMalloc((void **) & devY, ysize));
    gpu_error(hipMemcpy(devkwt, kwt, kwtsize, hipMemcpyHostToDevice));
    gpu_error(hipMalloc((void **) & devcutY, cutsize));
    // call the kernel function for precomputing
    precompute<<<och, ch>>>(och, ch, devkwt, devU);
    
    dim3 grid(bs, p, q);  // 3-D
    dim3 block(ch, 1, 1); // 1-D
    // call the kernel function for tiling
    tile<<<grid, block>>>(bs, p, q, ch, devin, devout, devsum, devY, devU, h, w, och, devfin);
    cudaSafeCall(hipGetLastError());

    dim3 cutgrid(bs*och, p, q);
    dim3 cutblock(1,1,1);



    cutpad<<<cutgrid, cutblock>>> (devY, devcutY, oph, opw);
    
    // copy from device to host.
    delete in;
    out = (float *)malloc(outsize);
    sum = (float *)malloc(sumsize);
    Y = (float *)malloc(ysize);
    cutY = (float *)malloc(cutsize);

    cudaSafeCall(hipMemcpy(out, devout, outsize, hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(sum, devsum, sumsize, hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(Y, devY, ysize, hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpy(cutY, devcutY, cutsize, hipMemcpyDeviceToHost));

    gpu_error(hipFree(devin));
    gpu_error(hipFree(devout));
    gpu_error(hipFree(devsum));
    gpu_error(hipFree(devkwt));
    gpu_error(hipFree(devU));
    gpu_error(hipFree(devfin));
    gpu_error(hipFree(devY));
    gpu_error(hipFree(devcutY));
}

int main(void) 
{
    auto engine = default_random_engine(0);
    auto rng = uniform_real_distribution<float>();
    int bs, ch, h, w, p, q, och, pad;
    
    bs = 2;
    ch = 2;
    h = 5;
    w = 5;
    och = 1;
    pad = 0;
    int insize = bs * ch * h * w * sizeof(float);
    int outsize, sumsize, ysize;
    float *in = new float[insize/sizeof(float)];
    float *t = in;
    float *out, *sum, *Y, *cutY;
    float *kernel_weights = new float[och*3*3*ch];
    int tsize = och*ch*3*3;
    float *tkw = kernel_weights;
    //put kernel weights
    LOOP(tsize)
    {
        tkw[ttsize] = 0;
    }
    tkw[0] = 1;
    tkw[9] = 1;
    //put input
    LOOP(bs)
    {
        LOOP(ch)
        {
            LOOP(h)
            {
                LOOP(w)
                {
                    *(t++) = rng(engine);
                }
            }
        }
    }
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(ch)
        {
            cout<<"{ ";
            LOOP(h)
            {
                cout<<"{ ";
                LOOP(w)
                {
                    cout<<in[((tbs*ch+tch)*h+th)*w+tw]<<" ";
                }
                cout<<"}\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
    cout<<"\nConvolving\n";
    int oph, opw;
    tilehost(och, ch, bs, h, w, pad, in, p, q, oph, opw, outsize, out, sumsize, sum, ysize, Y, cutY, kernel_weights);

    cout<<"\nConvolution finished\n\n";

      
    LOOP(bs)
    {
        cout<<"{ ";
        LOOP(och)
        {
            cout<<"{ ";
            LOOP(oph)
            {
                LOOP(opw)
                {
                    cout<<cutY[((tbs*och+toch)*oph+toph)*opw+topw]<<",";
                }
                cout<<";\n";
            }
            cout<<"}\n";
        }
        cout<<"}\n";
    }
    cout<<"}\n";
   

    // LOOP(bs)
    // {
    //     cout<<"{ ";
    //     LOOP(och)
    //     {
    //         cout<<"{ ";
    //         LOOP(p)
    //         {
    //             cout<<"{ ";
    //             LOOP(q)
    //             {
    //                 cout<<"{ ";
    //                 for(int i = 0; i < 2; i++)
    //                 {
    //                     for(int j = 0; j < 2; j++)
    //                     {
    //                         cout<<Y[((((tbs*och+toch)*p+tp)*q+tq)*2+i)*2+j]<<",";
    //                     }
    //                     cout<<";\n";
    //                 }
    //                 cout<<"}\n";
    //             }
    //             cout<<"}\n";
    //         }
    //         cout<<"}\n";
    //     }
    //     cout<<"}\n";
    // }
    return 0;
}