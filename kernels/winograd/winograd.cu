#include "hip/hip_runtime.h"
#include "wingheader.h"

#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)
#define cudaSafeCall(call)  \
        do {\
            hipError_t err = call;\
            if (hipSuccess != err) \
            {\
                std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__ << "): " \
                    << hipGetErrorString(err);\
                exit(EXIT_FAILURE);\
            }\
        } while(0)


void gpu_error(hipError_t const &code) {
    if(code != hipSuccess)
    {
        std::cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << std::endl;
        exit( EXIT_FAILURE );
    }
}

__global__ void precompute(int och, int ch, float* kernel_weights, float *U)
{
    int x = threadIdx.x;
    int bid = blockIdx.x;
    int offset = bid*ch + x;
   
    float g[4][3] = {
        {1, 0, 0},
        {0.5, 0.5, 0.5},
        {0.5, -0.5, 0.5},
        {0, 0, 1}
    };
    
    float g_t[3][4] ={
        {1, 0.5, 0.5, 0},
        {0, 0.5, -0.5, 0},
        {0, 0.5, 0.5, 1}
    };
    float *temp = (float *)malloc(3*4*sizeof(float));
    for(int i = 0; i <3; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                temp[i*4+j] += kernel_weights[offset*3*3+i*3+k] * g_t[k][j];
            }
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            U[offset*4*4+i*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                U[offset*4*4+i*4+j] += g[i][k] * temp[k*4+j];
            }
        }
    }
    free(temp);
}
__global__ void uv(int tch, int ch, float *devfin, float *U,  float *V)
{
    int x = threadIdx.x;
    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[x*4*4 + i*4 + j] = U[((x*ch+tch)*4+i)*4+j]*V[i*4+j];            
}
__global__ void amul(int tbs, int tp, int tq, int bs, int och, int p, int q, float *devsum, float *devY)
{
    float A_t[2][4] = {
        {1, 1, 1, 0},
        {0, 1, -1,-1}
    };
    float A[4][2] = {
        {1,0},
        {1,1},
        {1,-1},
        {0,-1}
    };
    int x = threadIdx.x;
    float *temp = (float *)malloc(2*4*sizeof(float));
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i*4+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i*4+j] += A_t[i][k] * devsum[((((tbs*och+x)*p+tp)*q+tq)*4+k)*4+j];
            }
        }
    }
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <2; ++j)
        {
            devY[((((tbs*och+x)*p+tp)*q+tq)*2+i)*2+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                devY[((((tbs*och+x)*p+tp)*q+tq)*2+i)*2+j] += temp[i*4+k] * A[k][j];
            }
        }
    }
    free(temp);
}

__global__ void paddev(float *devin, float *devinnopad, int h, int w, int pad)
{
    int newh = gridDim.y;
    int neww = gridDim.z;
    int tbsch = blockIdx.x;
    int tnewh = blockIdx.y;
    int tneww = blockIdx.z;
    int newhw = newh*neww;
    int hw = h*w;
    int th = tnewh-pad;
    int tw = tneww-pad;
    
    if(th >= 0 && th < h && tw >= 0 && tw < w)
        devin[tbsch*newhw + tnewh*neww + tneww] = devinnopad[tbsch*hw + th*w + tw];
    else
        devin[tbsch*newhw + tnewh*neww + tneww] = 0;
    
}

__global__ void cutpad(float  *devY, float *devcutY, int oph,int opw)
{
    int p = gridDim.y;
    int q = gridDim.z;
    int tbsch = blockIdx.x;
    int tp = blockIdx.y;
    int tq = blockIdx.z;
    //int newhw = newh*neww;
    //int pq4 = p*q*4;
    int ophopw = oph*opw;
    for(int i = 0; i < 2; i++)
    {
        for(int j = 0; j < 2; j++)
        {
            if(tp*2 + i < oph && tq*2 + j < opw)
                devcutY[tbsch*ophopw + (tp*2+i)*opw + (tq*2+j)] = devY[(((tbsch*p + tp)*q +tq)*2 + i)*2  + j];
        }
    }
}
    
__global__ void tile(int bs, int p, int q, int ch, float *devin, float *devout, float *devsum, float *devY, float *devU, int h, int w, int och, float *&devfin)
{
    float thrtile[4][4];    
    int tbs, tp, tq, tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    tch = threadIdx.x;
    // copy the tiles to thrtile
    int offset1 = (tbs*ch + tch)*h*w;
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
            thrtile[i][j] = devin[offset1 + th*w + tw];

    float B[4][4] = {
        {1,0,0,0},
        {0,1,-1,1},
        {-1,1,1,0},
        {0,0,0,-1}
    };
    float B_t[4][4] = {
        {1,0,-1,0},
        {0,1,1,0},
        {0,-1,1,0},
        {0,1,0,-1}
    };
    //Calculation of V
    float temp[4][4];
    float *V = (float *)  malloc(16*sizeof(float));
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += thrtile[i][k] * B[k][j];
            }   
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            V[i*4+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                V[i*4+j] += B_t[i][k] * temp[k][j];
            }
        }
    }
    float *fin = (float *)malloc(och*4*4*sizeof(float));
    uv<<<1,och>>>(tch, ch, fin, devU, V); 
    hipDeviceSynchronize();
    free(V);

    for(int toch = 0; toch<och; toch++)
        for(int i = 0; i < 4; i++)
            for(int j = 0; j < 4; j++)
               devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] = fin[(toch*4+i)*4+j];

    // sum along the channels, using log n summing
    free(fin);
    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+tch)*4 + i)*4 + j] += devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch+(tch+s))*4 + i)*4 + j];
        }
        __syncthreads();
    }
    if(tch == 0) 
    {
        LOOP(och)
            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devout[(((((tbs*och+toch)*p+tp)*q+tq)*ch)*4 + i)*4 + j];
    }
  if(tch == 0)
  {
      amul<<<1,och>>>(tbs, tp, tq, bs, och, p, q, devsum, devY);
      hipDeviceSynchronize();
  }
}
float * WING::forward(int och, int ch, int bs, int h, int w, int pad, float *in, int &oph, int &opw, float *kwt)
{
    float *devin, *devinnopad, *cutY;
    int insize = bs * ch * h * w * sizeof(float);
    int newh, neww;
 
    gpu_error(hipMalloc((void **) & devinnopad, insize));
    gpu_error(hipMemcpy(devinnopad, in, insize, hipMemcpyHostToDevice));

    newh = h + 2*pad;
    neww = w + 2*pad;
    oph = newh-2;
    opw = neww-2;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    insize = bs * ch * newh * neww * sizeof(float);
    gpu_error(hipMalloc((void **) & devin, insize));

    // call padding
    dim3 padgrid(bs*ch, newh, neww);
    dim3 padblock(1, 1, 1);
 
    paddev<<<padgrid,padblock>>>(devin, devinnopad, h, w, pad);

    gpu_error(hipFree(devinnopad));
    h = newh;
    w = neww;
    int p = max((h-2)/2, 0);
    int q = max((w-2)/2, 0);
    
    float *devout, *devsum, *devkwt, *devU, *devY, *devcutY;
    float *devfin;
    devout = devsum = nullptr;
 
    size_t kwtsize = och*ch*3*3*sizeof(float);
    size_t finsize = bs * p * q * ch * och * 4 * 4 * sizeof(float);
    size_t outsize = bs * och * p * q * ch * 4 * 4 * sizeof(float);
    size_t sumsize = bs * och * p * q * 4 * 4 * sizeof(float);
    size_t ysize = bs * och * p * q * 2 * 2 * sizeof(float);
    size_t usize = och*ch*4*4*sizeof(float);
    size_t cutsize = bs*och*oph*opw*sizeof(float);
 
    gpu_error(hipMalloc((void **) & devout, outsize));
    gpu_error(hipMalloc((void **) & devsum, sumsize));
    gpu_error(hipMalloc((void **) & devkwt, kwtsize));
    gpu_error(hipMalloc((void **) & devU, usize));
    gpu_error(hipMalloc((void **) & devfin, finsize));
    gpu_error(hipMalloc((void **) & devY, ysize));
    gpu_error(hipMemcpy(devkwt, kwt, kwtsize, hipMemcpyHostToDevice));
    gpu_error(hipMalloc((void **) & devcutY, cutsize));
    // call the kernel function for precomputing
    precompute<<<och, ch>>>(och, ch, devkwt, devU);
    gpu_error(hipFree(devkwt));
    
    dim3 grid(bs, p, q);  // 3-D
    dim3 block(ch, 1, 1); // 1-D
    // call the kernel function for tiling
    tile<<<grid, block>>>(bs, p, q, ch, devin, devout, devsum, devY, devU, h, w, och, devfin);
    cudaSafeCall(hipGetLastError());

    gpu_error(hipFree(devin));
    gpu_error(hipFree(devout));
    gpu_error(hipFree(devsum));
    
    gpu_error(hipFree(devU));
    gpu_error(hipFree(devfin));

    dim3 cutgrid(bs*och, p, q);
    dim3 cutblock(1,1,1);

    cutpad<<<cutgrid, cutblock>>> (devY, devcutY, oph, opw);   
    gpu_error(hipFree(devY));
    // copy from device to host.
    // delete in;
    cutY = (float *)malloc(cutsize);

    cudaSafeCall(hipMemcpy(cutY, devcutY, cutsize, hipMemcpyDeviceToHost));

    
    
    gpu_error(hipFree(devcutY));

    return cutY;
}
