#include "hip/hip_runtime.h"
//%%cuda --name winograd_mem.cu
#include "wingheader.h"

#define MAX_B 1
#define MAX_THREAD 1024
#define LOOP(x) for(int t##x = 0; t##x < x; t##x++)
#define cudaSafeCall(call)  \
        do {\
            hipError_t err = call;\
            if (hipSuccess != err) \
            {\
                std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__ << "): " \
                    << hipGetErrorString(err);\
                exit(EXIT_FAILURE);\
            }\
        } while(0)


void gpu_error(hipError_t const &code) {
    if(code != hipSuccess)
    {
        std::cerr << "GPUError: Code " << code << " : " << hipGetErrorString(code) << std::endl;
        exit( EXIT_FAILURE );
    }
}

__global__ void precompute(int och, int ch, float* kernel_weights, float *U)
{
    // int x = threadIdx.x;
    // int bid = blockIdx.x;
    // int offset = bid*ch + x;

    int tch = blockIdx.x;
    int toch = threadIdx.x;
   
    float g[4][3] = {
        {1, 0, 0},
        {0.5, 0.5, 0.5},
        {0.5, -0.5, 0.5},
        {0, 0, 1}
    };
    
    float g_t[3][4] ={
        {1, 0.5, 0.5, 0},
        {0, 0.5, -0.5, 0},
        {0, 0.5, 0.5, 1}
    };
    float temp[3][4];// = (float *)malloc(3*4*sizeof(float));
    for(int i = 0; i <3; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <3; ++k)
            {
                temp[i][j] += kernel_weights[((toch*ch + tch)*3 + i)*3+k] * g_t[k][j];
            }
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            U[((toch*ch + tch)*4 + i)*4+j] = 0;
            for(int k = 0; k <3; ++k)
            {
                U[((toch*ch + tch)*4 + i)*4+j] += g[i][k] * temp[k][j];
            }
        }
    }


    // int x = threadIdx.x;
    // int bid = blockIdx.x;
    // int offset = bid*ch + x;
   
    // float g[4][3] = {
    //     {1, 0, 0},
    //     {0.5, 0.5, 0.5},
    //     {0.5, -0.5, 0.5},
    //     {0, 0, 1}
    // };
    
    // float g_t[3][4] ={
    //     {1, 0.5, 0.5, 0},
    //     {0, 0.5, -0.5, 0},
    //     {0, 0.5, 0.5, 1}
    // };
    // float temp[3][4];// = (float *)malloc(3*4*sizeof(float));
    // for(int i = 0; i <3; ++i)
    // {
    //     for(int j = 0; j <4; ++j)
    //     {
    //         temp[i][j] = 0;
    //         for(int k = 0; k <3; ++k)
    //         {
    //             temp[i][j] += kernel_weights[offset*3*3+i*3+k] * g_t[k][j];
    //         }
    //     }
    // }
    // for(int i = 0; i <4; ++i)
    // {
    //     for(int j = 0; j <4; ++j)
    //     {
    //         U[offset*4*4+i*4+j] = 0;
    //         for(int k = 0; k <3; ++k)
    //         {
    //             U[offset*4*4+i*4+j] += g[i][k] * temp[k][j];
    //         }
    //     }
    // }
    // free(temp);
}


__global__ void paddev(float *devin, float *devinnopad, int h, int w, int pad)
{
    int newh = gridDim.y;
    int neww = gridDim.z;
    int tbs = blockIdx.x;
    int tch = threadIdx.x;
    int ch = blockDim.x;
    int tnewh = blockIdx.y;
    int tneww = blockIdx.z;
    int newhw = newh*neww;
    int hw = h*w;
    int th = tnewh-pad;
    int tw = tneww-pad;
    int tbsch = tbs*ch + tch;
    
    if(th >= 0 && th < h && tw >= 0 && tw < w)
        devin[tbsch*newhw + tnewh*neww + tneww] = devinnopad[tbsch*hw + th*w + tw];
    else
        devin[tbsch*newhw + tnewh*neww + tneww] = 0;
    
}

__global__ void cutpad(float  *devY, float *devcutY, int oph,int opw)
{
    int p = gridDim.y;
    int q = gridDim.z;
    int tbs = blockIdx.x;
    int tp = blockIdx.y;
    int tq = blockIdx.z;
    int toch = threadIdx.x;
    int och = blockDim.x;
    int offset = tbs*och+toch;
    //int newhw = newh*neww;
    //int pq4 = p*q*4;
    int ophopw = oph*opw;
    for(int i = 0; i < 2; i++)
    {
        for(int j = 0; j < 2; j++)
        {
            if(tp*2 + i < oph && tq*2 + j < opw)
                devcutY[offset*ophopw + (tp*2+i)*opw + (tq*2+j)] = devY[(((offset*p + tp)*q +tq)*2 + i)*2  + j];
        }
    }
}

__global__ void tile(int bs, int p, int q, int ch, float *devin, float *devsum, float *devU, int h, int w, int och, float *devfin)
{
    float thrtile[4][4];    
    int tbs, tp, tq, tch, Tch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    Tch = threadIdx.x;
    float V[4][4];// = (float *)  malloc(16*sizeof(float));
    // if(Tch%och==0)
    // {
    tch = Tch / och;
    // copy the tiles to thrtile
    int offset1 = (tbs*ch + tch)*h*w;
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
            thrtile[i][j] = devin[offset1 + th*w + tw];

    float B[4][4] = {
        {1,0,0,0},
        {0,1,-1,1},
        {-1,1,1,0},
        {0,0,0,-1}
    };
    float B_t[4][4] = {
        {1,0,-1,0},
        {0,1,1,0},
        {0,-1,1,0},
        {0,1,0,-1}
    };
    //Calculation of V
    float temp[4][4];

    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += thrtile[i][k] * B[k][j];
            }   
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            V[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                V[i][j] += B_t[i][k] * temp[k][j];
            }
        }
    }
    // }
    __syncthreads();

    int toch = Tch % och;
    tch = Tch / och;

    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] = devU[((toch*ch+tch)*4+i)*4+j]*V[i][j]; 
    
    __syncthreads();

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            toch = Tch % och;
            // LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] += devfin[(((((tbs*p+tp)*q+tq)*ch+(tch+s))*och+toch)*4+i)*4+j];
        }
        __syncthreads();
    }

    if(tch == 0) 
    {

            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devfin[(((((tbs*p+tp)*q+tq)*ch+0)*och+toch)*4+i)*4+j];
    }
     __syncthreads();
  
}

__global__ void tile2(int bs, int p, int q, int ch, float *devin, float *devsum, float *devU, int h, int w, int och, float *devfin)
{
    float thrtile[4][4];    
    int tbs, tp, tq, tch, tbsf, x;
    tbsf = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    x = threadIdx.x;
    tbs = tbsf%bs;

    int och_pb = MAX_THREAD/ch;
    int tf = tbsf / bs;
    int toch = x/ch + tf*(och_pb);
    tch = x%ch; 

    float V[4][4];// = (float *)  malloc(16*sizeof(float));
    // if(Tch%och==0)
    // {
   // tch = Tch / och;
    // copy the tiles to thrtile
    int offset1 = (tbs*ch + tch)*h*w;
    for(int th = 2*tp, i = 0; i < 4; th++, i++)
        for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
            thrtile[i][j] = devin[offset1 + th*w + tw];

    float B[4][4] = {
        {1,0,0,0},
        {0,1,-1,1},
        {-1,1,1,0},
        {0,0,0,-1}
    };
    float B_t[4][4] = {
        {1,0,-1,0},
        {0,1,1,0},
        {0,-1,1,0},
        {0,1,0,-1}
    };
    //Calculation of V
    float temp[4][4];

    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += thrtile[i][k] * B[k][j];
            }   
        }
    }
    for(int i = 0; i <4; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            V[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                V[i][j] += B_t[i][k] * temp[k][j];
            }
        }
    }
    // }
    __syncthreads();

   // int toch = Tch % och;
    //tch = Tch / och;

    for(int i = 0; i <4; ++i)
        for(int j = 0; j <4; ++j)
            devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] = devU[((toch*ch+tch)*4+i)*4+j]*V[i][j]; 
    
    __syncthreads();

    for(int s = 1; s < ch; s *= 2)
    {
        if(tch % (2*s) == 0 && tch+s < ch)
        {
            //toch = Tch % och;
            // LOOP(och)
                for(int i = 0; i < 4; i++)
                    for(int j = 0; j < 4; j++)
                        devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] += devfin[(((((tbs*p+tp)*q+tq)*ch+(tch+s))*och+toch)*4+i)*4+j];
        }
        __syncthreads();
    }

    if(tch == 0) 
    {

            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devfin[(((((tbs*p+tp)*q+tq)*ch+0)*och+toch)*4+i)*4+j];
    }
     __syncthreads();
  
}
    
// __global__ void tile(int bs, int p, int q, int ch, float *devin, float *devsum, float *devU, int h, int w, int och, float *devfin)
// {
//     float thrtile[4][4];    
//     int tbs, tp, tq, tch, tbsoch, toch;
//     tbsoch = blockIdx.x;
//     tp = blockIdx.y;
//     tq = blockIdx.z;
//     tch = threadIdx.x;
//     tbs  = tbsoch/och;
//     toch = tbsoch%och;
//     float V[4][4];// = (float *)  malloc(16*sizeof(float));
//     // if(Tch%och==0)
//     // {
//     //tch = Tch / och;
//     // copy the tiles to thrtile
//     int offset1 = (tbs*ch + tch)*h*w;
//     for(int th = 2*tp, i = 0; i < 4; th++, i++)
//         for(int tw = 2*tq, j = 0; j < 4; tw++, j++)
//             thrtile[i][j] = devin[offset1 + th*w + tw];

//     float B[4][4] = {
//         {1,0,0,0},
//         {0,1,-1,1},
//         {-1,1,1,0},
//         {0,0,0,-1}
//     };
//     float B_t[4][4] = {
//         {1,0,-1,0},
//         {0,1,1,0},
//         {0,-1,1,0},
//         {0,1,0,-1}
//     };
//     //Calculation of V
//     float temp[4][4];

//     for(int i = 0; i <4; ++i)
//     {
//         for(int j = 0; j <4; ++j)
//         {
//             temp[i][j] = 0;
//             for(int k = 0; k <4; ++k)
//             {
//                 temp[i][j] += thrtile[i][k] * B[k][j];
//             }   
//         }
//     }
//     for(int i = 0; i <4; ++i)
//     {
//         for(int j = 0; j <4; ++j)
//         {
//             V[i][j] = 0;
//             for(int k = 0; k <4; ++k)
//             {
//                 V[i][j] += B_t[i][k] * temp[k][j];
//             }
//         }
//     }
//     // }
//     __syncthreads();

//     //int toch = Tch % och;
//     //tch = Tch / och;

//     for(int i = 0; i <4; ++i)
//         for(int j = 0; j <4; ++j)
//             devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] = devU[((toch*ch+tch)*4+i)*4+j]*V[i][j]; 
    
//     __syncthreads();

//     for(int s = 1; s < ch; s *= 2)
//     {
//         if(tch % (2*s) == 0 && tch+s < ch)
//         {
//             //toch = Tch % och;
//             // LOOP(och)
//                 for(int i = 0; i < 4; i++)
//                     for(int j = 0; j < 4; j++)
//                         devfin[(((((tbs*p+tp)*q+tq)*ch+tch)*och+toch)*4+i)*4+j] += devfin[(((((tbs*p+tp)*q+tq)*ch+(tch+s))*och+toch)*4+i)*4+j];
//         }
//         __syncthreads();
//     }

//     if(tch == 0) 
//     {

//             for(int i = 0; i < 4; i++)
//                 for(int j = 0; j < 4; j++)
//                     devsum[((((tbs*och+toch)*p+tp)*q+tq)*4 + i)*4 + j] = devfin[(((((tbs*p+tp)*q+tq)*ch+0)*och+toch)*4+i)*4+j];
//     }
//      __syncthreads();
  
// }

__global__ void lastcal(int och, int p, int q, int bs, float *devsum, float *devY)
{
    int tbs, tp, tq, toch;
    tbs = blockIdx.x;
    tp = blockIdx.y;
    tq = blockIdx.z;
    toch = threadIdx.x;

  float A_t[2][4] = {
        {1, 1, 1, 0},
        {0, 1, -1,-1}
    };
    float A[4][2] = {
        {1,0},
        {1,1},
        {1,-1},
        {0,-1}
    };
    // int x = 0; //threadIdx.x;
    float temp[2][4];// = (float *)malloc(2*4*sizeof(float));
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <4; ++j)
        {
            temp[i][j] = 0;
            for(int k = 0; k <4; ++k)
            {
                temp[i][j] += A_t[i][k] * devsum[((((tbs*och+toch)*p+tp)*q+tq)*4+k)*4+j];
            }
        }
    }
    for(int i = 0; i <2; ++i)
    {
        for(int j = 0; j <2; ++j)
        {
            devY[((((tbs*och+toch)*p+tp)*q+tq)*2+i)*2+j] = 0;
            for(int k = 0; k <4; ++k)
            {
                devY[((((tbs*och+toch)*p+tp)*q+tq)*2+i)*2+j] += temp[i][k] * A[k][j];
            }
        }
    }
}


float * WING::forward(int och, int ch, int bs, int h, int w, int pad, float *in, int &oph, int &opw, float *kwt)
{
    float *devin, *devinnopad, *cutY, *devkwt, *devU;
    size_t insize = bs * ch * h * w * sizeof(float);
    int newh, neww;
 
    gpu_error(hipMalloc((void **) & devinnopad, insize));
    gpu_error(hipMemcpy(devinnopad, in, insize, hipMemcpyHostToDevice));

    newh = h + 2*pad;
    neww = w + 2*pad;
    oph = newh-2;
    opw = neww-2;
    if(newh%2)
        newh++;
    if(neww%2)
        neww++;
    if(newh < 4)
        newh = 4;
    if(neww < 4)
        neww = 4;

    insize = bs * ch * newh * neww * sizeof(float);
    gpu_error(hipMalloc((void **) & devin, insize));

    // call padding
    dim3 padgrid(bs, newh, neww);
    dim3 padblock(ch, 1, 1);
 
    paddev<<<padgrid,padblock>>>(devin, devinnopad, h, w, pad);
    gpu_error(hipFree(devinnopad));
    h = newh;
    w = neww;

    size_t kwtsize = och*ch*3*3*sizeof(float);    
    size_t usize = och*ch*4*4*sizeof(float);
    gpu_error(hipMalloc((void **) & devkwt, kwtsize));
    gpu_error(hipMalloc((void **) & devU, usize));
    gpu_error(hipMemcpy(devkwt, kwt, kwtsize, hipMemcpyHostToDevice));
    precompute<<<ch, och>>>(och, ch, devkwt, devU);
    gpu_error(hipFree(devkwt));

    size_t cutsize = bs*och*oph*opw*sizeof(float);
    cutY = (float *)malloc(cutsize);

    float *devsum, *devY, *devcutY;
    float *devfin;
    //devout = devsum = nullptr;
     int p = max((h-2)/2, 0);
    int q = max((w-2)/2, 0);

    //size_t finsize = bs * p * q * ch * och * 4 * 4 * sizeof(float);
    size_t finsize = MAX_B * p * q * ch * och * 4 * 4 * sizeof(float);
   // size_t outsize = bs * och * p * q * ch * 4 * 4 * sizeof(float);
    size_t sumsize = bs * och * p * q * 4 * 4 * sizeof(float);
    size_t ysize = bs * och * p * q * 2 * 2 * sizeof(float);

 
    //gpu_error(hipMalloc((void **) & devout, outsize));
    gpu_error(hipMalloc((void **) & devsum, sumsize));

    gpu_error(hipMalloc((void **) & devfin, finsize));
    // printf("%d %d %d\n", insize, sumsize, finsize);

    // call the kernel function for precomputing
    
    
    // dim3 grid(bs, p, q);  // 3-D
     // 1-D
    // // call the kernel function for tiling
    // tile<<<grid, block>>>(bs, p, q, ch, devin, devsum, devU, h, w, och, devfin);

    // gpu_error(hipFree(devfin));
    // gpu_error(hipFree(devin));    
    // gpu_error(hipFree(devU));

    // // cudaSafeCall(hipGetLastError());

    // //gpu_error(hipFree(devout));
    // dim3 block2(och, 1, 1);
    // gpu_error(hipMalloc((void **) & devY, ysize));
    // lastcal<<<grid,block2>>>(och, p, q, bs, devsum, devY);
 
    // __global__ float * t_devin = devin;
    // __global__ float * t_devsum = devsum;
    size_t binsize = ch * newh * neww ;
    size_t dsumsize = och * p * q * 4 * 4 ;
    int bsg = (bs+MAX_B-1)/MAX_B;
    int prevb = 0;
    LOOP(bsg)
    {
        int currb = MAX_B;
        if(tbsg == bsg-1 && bs % MAX_B != 0)
            currb = bs % MAX_B;
        //printf("%d %d\n", currb, tbsg);
        if(och*ch <= MAX_THREAD)
        {
            dim3 grid(currb, p, q); 
            dim3 block(och*ch, 1, 1);
            tile<<<grid, block>>>(currb, p, q, ch, devin + prevb*binsize, devsum + prevb*dsumsize, devU, h, w, och, devfin);
        }
        else
        {
            int f = (och*ch)/MAX_THREAD;
            dim3 grid(currb*f, p, q); 
            dim3 block(MAX_THREAD, 1, 1);
            tile2<<<grid, block>>>(currb, p, q, ch, devin + prevb*binsize, devsum + prevb*dsumsize, devU, h, w, och, devfin);   
        }
        // t_devin += currb * binsize;
        // t_devsum += currb * dsumsize;
        prevb  += currb;
    }

    gpu_error(hipFree(devfin));
    gpu_error(hipFree(devin));    
    gpu_error(hipFree(devU));

    //gpu_error(hipFree(devout));
    dim3 grid2(bs, p, q);
    dim3 block2(och, 1, 1);
    gpu_error(hipMalloc((void **) & devY, ysize));
    lastcal<<<grid2,block2>>>(och, p, q, bs, devsum, devY);
    gpu_error(hipFree(devsum));

    dim3 cutgrid(bs, p, q);
    dim3 cutblock(och,1,1);
    
    
    gpu_error(hipMalloc((void **) & devcutY, cutsize));
    cutpad<<<cutgrid, cutblock>>> (devY, devcutY, oph, opw);   
    gpu_error(hipFree(devY));

    cudaSafeCall(hipMemcpy(cutY, devcutY, cutsize, hipMemcpyDeviceToHost));
    
    gpu_error(hipFree(devcutY));
  
    return cutY;

}